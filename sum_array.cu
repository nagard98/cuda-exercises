
#include <hip/hip_runtime.h>
#include<stdio.h>

#define N 100
#define threadsPerBlock 64
#define numBlocks 2

__global__ void sum(int* a, int* b, int* c){
    int tid = blockDim.x * blockIdx.x + threadIdx.x;
    if(tid < N) c[tid] = a[tid] + b[tid];
}

int main(void){
    int a[N], b[N], c[N];
    int *a_dev, *b_dev, *c_dev;

    //Pageable memory
    //cudaMalloc((void**)&a_dev, N * sizeof(int));
    //cudaMalloc((void**)&b_dev, N * sizeof(int));
    //cudaMalloc((void**)&c_dev, N * sizeof(int));

    //Pinned memory
    hipHostAlloc((void**)&a_dev, N * sizeof(int), hipHostMallocDefault);
    hipHostAlloc((void**)&b_dev, N * sizeof(int), hipHostMallocDefault);
    hipHostAlloc((void**)&c_dev, N * sizeof(int), hipHostMallocDefault);

    for(int i=0; i<N; i++){
        a[i] = i;
        b[i] = i * i;
    }

    hipMemcpy(a_dev, a, N * sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(b_dev, b, N * sizeof(int), hipMemcpyHostToDevice);

    sum<<<numBlocks, threadsPerBlock>>>(a_dev, b_dev, c_dev);
    
    hipMemcpy(c, c_dev, N * sizeof(int), hipMemcpyDeviceToHost);

    for(int i=0; i<N; i++){
        printf("%d + %d = %d \n", a[i], b[i], c[i]);
    }

    hipHostFree(a_dev);
    hipHostFree(b_dev);
    hipHostFree(c_dev);

    return 0;

}