
#include <hip/hip_runtime.h>
#include<stdio.h>

#define N 100
#define threadsPerBlock 64
#define numBlocks 2

__global__ void sum(int* a, int* b, int* c){
    int tid = blockDim.x * blockIdx.x + threadIdx.x;
    if(tid < N) c[tid] = a[tid] + b[tid];
}

int main(void){
    int *a, *b, *c;
    int *a_dev, *b_dev, *c_dev;

    //Mapped memory
    hipHostAlloc((void**)&a, N * sizeof(int), hipHostMallocMapped);
    hipHostAlloc((void**)&b, N * sizeof(int), hipHostMallocMapped);
    hipHostAlloc((void**)&c, N * sizeof(int), hipHostMallocMapped);

    for(int i=0; i<N; i++){
        a[i] = i;
        b[i] = i * i;
    }

    hipHostGetDevicePointer(&a_dev, a, 0);
    hipHostGetDevicePointer(&b_dev, b, 0);
    hipHostGetDevicePointer(&c_dev, c, 0);

    sum<<<numBlocks, threadsPerBlock>>>(a_dev, b_dev, c_dev);

    for(int i=0; i<N; i++){
        printf("%d + %d = %d \n", a[i], b[i], c[i]);
    }

    hipHostFree(a_dev);
    hipHostFree(b_dev);
    hipHostFree(c_dev);

    return 0;

}