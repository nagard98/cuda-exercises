
#include <hip/hip_runtime.h>
#include<stdio.h>

__global__ void miokernel(void){
	int tid = blockIdx.x * blockDim.x + threadIdx.x;
	printf("Sono il thread %d!\n", tid);
}

int main(void){
	miokernel<<<2,32>>>();
  	hipDeviceSynchronize();
	printf("Hello, World!\n");
	return 0;
}